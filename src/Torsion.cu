/*
 * Torsion.cpp
 *
 *  Created on: 31/08/2019
 *      Author: alfonso
 */


#include "Torsion.hpp"

Torsion::Torsion(int avance, int paso, vector<EntradaR> &vrec,
		vector<EntradaD> &dicc, vector<string> &vscl) :
		vrec(vrec), vdiccionario(dicc), vscl(vscl), idc(0), pdm_host(nullptr) {
	this->avance = avance;
	this->paso = paso;
}

/**
 *
 */
void Torsion::calculaTorsion() {
	size_t tam = vrec.size();

	for (size_t i = avance; i < tam; i += paso) {

		size_t tamo = vrec[i].voracion.size();
		for (size_t j = 0; j < tamo; j++) {
			vector<string> vcon;
			string &sidea = vrec[i].voracion[j];
			remueveCarL(sidea);
			split(vcon, sidea, " ");
			vector<size_t> vaux;
			for (string scon : vcon) {

				size_t id = concepto2id(scon);
				vaux.push_back(id);

			}
			vrec[i].vvidc.push_back(vaux);
		}
	}

	cout << "Total de conceptos empleados (vectores): " << idc << " ("
			<< idc * sizeof(float) * TAMV << ") bytes" << endl;

	cout << "Total de recursos (vector torsion): " << vrec.size() << " ("
			<< vrec.size() * sizeof(float) * TAMV << ") bytes" << endl;

	calculaTorsionC();
}

/**
 * Metodo que llama al codigo de CUDA para el calculo de la Torsion, aloja memoria y genera los
 * vectores necesarios tanto en Host como en Device
 */
void Torsion::calculaTorsionC() {
	//diccionario
	pdm_host = new float[idc * TAMV];
	generaDiccionarioMin();

	//vectores de torsion (resultados)
	prec_host = new float[vrec.size() * TAMV];



	//array que define la entradas de los recursos inicio y fin (posicion de las oraciones)
	this->pentradarmin_host = new EntradaRmin[vrec.size()];

	size_t tam_total=this->generaOraciones();

	cout << "Tam total size_t: "<<tam_total<<" reportado por oraciones: "<<sizeof(this->proracion_host)<<endl;


	// Se definen memoria para el dispositivo

	float *pdm_device= nullptr;


	//se aloja la memoria en el dispositivo
	hipMalloc((void**) &(pdm_device), idc*TAMV * sizeof(float));




	//se libera la memoria en el dispositivo
	hipFree(pdm_device);


	//se libera memoria en el host
	for(size_t i=0;i<this->num_oraciones;i++){
		delete[] this->proracion_host[i];
	}
	delete[] this->proracion_host;

	delete[] this->pentradarmin_host;


	delete[] prec_host;
	delete[] pdm_host;
}

/**
 *
 * @param theStringVector
 * @param theString
 * @param theDelimiter
 */
void Torsion::split(vector<string> &theStringVector, const string &theString,
		const string &theDelimiter) {
	size_t start = 0, end = 0;

	while (end != string::npos) {
		end = theString.find(theDelimiter, start);

		// If at end, use length=maxLength.  Else use length=end-start.
		theStringVector.push_back(
				theString.substr(start,
						(end == string::npos) ? string::npos : end - start));

		// If at end, use start=maxSize.  Else use start=end+delimiter.
		start = (
				(end > (string::npos - theDelimiter.size())) ?
						string::npos : end + theDelimiter.size());
	}
}

Torsion::~Torsion() {

}


/**
 *
 * @param dicc
 * @param scad
 * @return
 */
size_t Torsion::concepto2id(string &scad) {

	boost::trim_right(scad);
	boost::trim_left(scad);

	if (scad.size() > 0) {

		for (EntradaD &e : vdiccionario) {

			if (e.concepto == scad) {
				if (e.idc == -1) {
					e.idc = idc;
					idc++;
				}
				return e.idc;
			}
		}
	}

	return -1;
}


/**
 * @see https://stackoverflow.com/questions/20326356/how-to-remove-all-the-occurrences-of-a-char-in-c-string
 * @param scad
 * @return
 */
void Torsion::remueveCarL(string &scad) {
	for (auto sb : vscl) {
		boost::replace_all(scad, sb, " ");
	}
}

/**
 *
 */
void Torsion::generaDiccionarioMin() {
	cout <<"Se genera el diccionario minimo"<<endl;
	for (auto &vaux : vdiccionario) {
		if (vaux.idc >= 0) {
			for (size_t i = 0; i < TAMV; i++) {
				*(pdm_host + TAMV * vaux.idc + i) = vaux.v[i];
			}
			//cout << vaux.idc<<": "<<vaux.concepto<<endl;
		}
	}
}

size_t Torsion::generaOraciones() {

	num_oraciones = 0;

	for (auto rec_aux : vrec) {
		num_oraciones += rec_aux.voracion.size();
	}

	this->proracion_host = new size_t*[num_oraciones];

	size_t tam_total=0;
	size_t tam_vrec = vrec.size();
	size_t i = 0;
	for (size_t k = 0; k < tam_vrec; k++) {
		(this->pentradarmin_host + k)->inicio = i;
		for (auto v1 : vrec[k].vvidc) {

			size_t tam = v1.size();
			tam_total+=tam;
			this->proracion_host[i] = new size_t[tam];
			for (size_t j = 0; j < tam; j++) {
				this->proracion_host[i][j] = v1[j];
			}
			(this->pentradarmin_host + k)->fin = i;
			i++;
		}

	}

	return tam_total;
}

