#include "hip/hip_runtime.h"



#include "Comun.hpp"


/**
 * Funcion que calcula el valor de tensor de Levi-Civita (en espacios de dimensionalidad arbitraria)
 * https://en.wikipedia.org/wiki/Levi-Civita_symbol
 * https://www.johndcook.com/blog/2018/09/16/permutation-tensor/
 * @param i
 * @param j
 * @param k
 * @return
 */
__device__ float klcfat(float i,float j,float k){

    if (i == j || j == k || k == i) {
        return 0.0;
    }
    float aux = (j - i) * (k - i) * (k - j);

    return aux / fabs(aux);
}

/**
 *
 */
__global__ void kcalculaTorsion(){

}
